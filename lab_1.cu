
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void parallel_vector_add(int* d_a, int* d_b, int* d_c, int* d_n) {
	int i = (blockIdx.x*blockDim.x) + threadIdx.x;
	// printf("I am thread #%d\n", i);
	if(i < *d_n) {
		printf("I am thread #%d. and about to computer c[%d].\n", i, i);
		d_c[i] = d_a[i]+d_b[i];
	} else {
		printf("I am thread #%d and doing nothing.\n", i);
	}
}

int main() {
	// allocate and initialize host memory
	int n;
	scanf("%d", &n);

	int h_a[n];
      	int h_b[n];
	int h_c[n];

	for(int i = 0; i < n; i++) {
		h_a[i] = i;
		h_b[i] = n-1;
	}

	// Part 1: Copy data from host to device
	// allocate device memory for a, b, and c
	int *d_a, *d_b, *d_c, *d_n;
	// copy a and b to device memory
	hipMalloc((void **) &d_a, n*sizeof(int));
	hipMalloc((void **) &d_b, n*sizeof(int));
	hipMalloc((void **) &d_c, n*sizeof(int));
	hipMalloc((void **) &d_n, sizeof(int));
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(d_a, &h_a, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &h_b, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

	// Part 2: Kernel launch
	// kernel launch code which let the device performs the actual vector addtion
	int numB;	// number of block
	numB = n%512 ? (n/512)+1 : n/512;	// 512 is a number of thread per block.

	hipEventRecord(start);
	// 512 is a number of thread per block which can be change without a problem.
	parallel_vector_add<<<numB, 512>>>(d_a, d_b, d_c, d_n);
	hipEventRecord(stop);
	hipDeviceSynchronize();

	// Part 3: Copy data from device back to host, and free all data allocate on device
	hipMemcpy(&h_c, d_c, n*sizeof(int), hipMemcpyDeviceToHost);
	
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	for(int i = 0; i < n; i++) {
		printf("%d ", h_c[i]);
	}
	printf("\n Time used: %f milliseconds\n", milliseconds);
}
